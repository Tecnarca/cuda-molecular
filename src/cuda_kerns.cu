#include "hip/hip_runtime.h"
#ifndef DEBUG
#define INSIZE 256 //dimension of 'in'. Used items: 192 (64*3)
#define N_ATOMS 64 //atoms inside 'in'
#define N_FRAGS 4 //number of fragments
#define DEBUG 1 //should we execute the debug prints and checks?
#define MASKSIZE 256 //dimension of the 'mask'
#define VOLUMESIZE 1000000 //dimension of 'score_pos'
#define MAX_ANGLE 256 //up to which angle we need to run the algorithm?
#define LIMIT_DISTANCE2 2.0 //used in fragment_is_bumping, it is the minimum distance between to atoms
#define GRID_FACTOR_D 0.5
#define PI 3.141592653589793238462643383279
#define RADIAN_COEF PI/128.0;
#endif
#include <cuda_kerns.h>
#include <stdio.h>
#include <chrono>

texture<float, 1, hipReadModeElementType> texScore_pos;
texture<int, 1, hipReadModeElementType> texMask;

__inline__ __device__ int warpReduce(int val) {
	
	for (int i = warpSize/2; i > 0; i/=2){
		val += __shfl_down_sync(0xffffffff, val, i, 32);
	}
	
	return val;
}

__inline__ __device__ int blockReduce(int val) {
	
	static __shared__ int shared[32];

	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;

	val = warpReduce(val);

	if (lane==0) shared[wid]=val;
	
	__syncthreads();

	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
	
	if (wid==0) val = warpReduce(val);
	
	return val;
}

__device__ void compute_matrix( const int rotation_angle,
								const float x_orig, const float y_orig, const float z_orig,
								const float x_vector, const float y_vector, const float z_vector, float* matrix){

	const float u = (float)x_vector - x_orig;
	const float v = (float)y_vector - y_orig;
	const float w = (float)z_vector - z_orig;
	const float u2 = u * u;
	const float v2 = v * v;
	const float w2 = w * w;

	const float l2 = u * u + v * v + w * w;
	const float l = sqrtf(l2);

	const float angle_rad = (float)-rotation_angle*RADIAN_COEF;
	const float sint = sin(angle_rad);
	const float cost = cos(angle_rad);
	const float one_minus_cost = (float)1.0 - cost;

	matrix[0] =	(u2 + (v2 + w2) * cost) / l2;
	matrix[1] =	(u* v * one_minus_cost - w* l * sint) / l2;
	matrix[2] =	(u* w * one_minus_cost + v* l * sint) / l2;
	matrix[3] =	((x_orig * (v2 + w2) - u * (y_orig * v + z_orig * w)) * one_minus_cost + (y_orig * w - z_orig * v) * l * sint) / l2;

	matrix[4] =	(u* v * one_minus_cost + w* l * sint) / l2;
	matrix[5] =	(v2 + (u2 + w2) * cost) / l2;
	matrix[6] =	(v* w * one_minus_cost - u* l * sint) / l2;
	matrix[7] =	((y_orig * (u2 + w2) - v * (x_orig * u + z_orig * w)) * one_minus_cost + (z_orig * u - x_orig * w) * l * sint) / l2;

	matrix[8] =	(u* w * one_minus_cost - v* l * sint) / l2;
	matrix[9] =	(v* w * one_minus_cost + u* l * sint) / l2;
	matrix[10]=	(w2 + (u2 + v2) * cost) / l2;
	matrix[11]=	((z_orig * (u2 + v2) - w * (x_orig * u + y_orig * v)) * one_minus_cost + (x_orig * v - y_orig * u) * l * sint) / l2;
}

__global__ void rotate(float* in, int* mask, int iter, float precision, int* start, int* stop){
	
	const int index = blockIdx.x;
	const int curr_start = start[iter];
	const int curr_stop = stop[iter];
	const int x = threadIdx.x;
	const int y = threadIdx.x + N_ATOMS;
	const int z = threadIdx.x + 2*N_ATOMS;
	const int offset = ceil(index*INSIZE/precision);

	//This can be shared and computed only once per iter instance of rotate<<<,>>>() instead that once per thread!
	//but how to do it efficently? Shared memory? Then, how do i initialize it?
	float m[12];

	__shared__ float in_s[N_ATOMS*3];

	in_s[x] = in[x];
	in_s[y] = in[y];
	in_s[z] = in[z];

	__syncthreads();

	compute_matrix(index*precision,in_s[curr_start],in_s[curr_start+N_ATOMS],in_s[curr_start+2*N_ATOMS],in_s[curr_stop],in_s[curr_stop+N_ATOMS], in_s[curr_stop+2*N_ATOMS], m);

	//is this line correct? Can we optimize this access with a 2D texture of dimension (64,4)? (probably no)
	/*The line IS NOT correct! causes a memory error, detectable only by calling
	cudaMemoryTest() before the call of rotate. Not even cuda-memcheck catched it! We revert temporarly to standard non-texturized array...*/
	const int mask_x = mask[x+iter*N_ATOMS];/*tex1Dfetch(texMask, x+iter*N_ATOMS);*/

	if(mask_x == 1){
		in[x+offset] = m[0] * in_s[x] + m[1] * in_s[y] + m[2] * in_s[z] + m[3];
		in[y+offset] = m[4] * in_s[x] + m[5] * in_s[y] + m[6] * in_s[z] + m[7];
		in[z+offset] = m[8] * in_s[x] + m[9] * in_s[y] + m[10] * in_s[z] + m[11];
	} else {
		in[x+offset]=in_s[x];
		in[y+offset]=in_s[y];
		in[z+offset]=in_s[z];		
	}
}

__global__ void measure_shotgun (float* in, float* scores, int* shotgun, float precision, int iter){
	const int index = blockIdx.x;
	const int writers = threadIdx.x;
	const int x = threadIdx.x + index*INSIZE;
	const int y = threadIdx.x + index*INSIZE + N_ATOMS;
	const int z = threadIdx.x + index*INSIZE + 2*N_ATOMS;

	int index_x = (int) (in[x]*GRID_FACTOR_D);
	int index_y = (int) (in[y]*GRID_FACTOR_D);
	int index_z = (int) (in[z]*GRID_FACTOR_D);

	if(threadIdx.x==0) shotgun[index] = 0;
	__syncthreads();

	if (index_x < 0) index_x = 0;
	if (index_x > 100) index_x = 100;
	if (index_y < 0) index_y = 0;
	if (index_y > 100) index_y = 100;
	if (index_z < 0) index_z = 0;
	if (index_z > 100) index_z = 100;

	//Is this line correct? Can we optimize this access pattern with a 3D texture of dimension (100,100,100)? (probably yes)
	int score = scores[index_x+100*index_y+10000*index_z];/*tex1Dfetch(texScore_pos, index_x+100*index_y+10000*index_z);*/

	int reduced = blockReduce(score);
	if(!writers) shotgun[index] = reduced;
}

__global__ void fragment_is_bumping(float* in, int* mask, int* is_bumping_p, int iter, float precision){
	const int index = blockIdx.y;
	int ix = threadIdx.x;
	int jx = blockIdx.x;
	int iy = threadIdx.x + N_ATOMS;
	int jy = blockIdx.x + N_ATOMS;
	int iz = threadIdx.x + 2*N_ATOMS;
	int jz = blockIdx.x + 2*N_ATOMS;
	int offset = index*INSIZE;

	__shared__ float in_s[N_ATOMS*3];

	in_s[ix] = in[ix+offset];
	in_s[iy] = in[iy+offset];
	in_s[iz] = in[iz+offset];

	__syncthreads();

	const float diff_x = in_s[ix] - in_s[jx];
	const float diff_y = in_s[iy] - in_s[jy];
	const float diff_z = in_s[iz] - in_s[jz];
	const float distance2 = diff_x * diff_x +  diff_y * diff_y +  diff_z * diff_z;

	//Are these lines correct?
	int m_ix = mask[ix+iter*N_ATOMS];/*tex1Dfetch(texMask, ix+iter*N_ATOMS);*/
	int m_jx = mask[jx+iter*N_ATOMS];/*tex1Dfetch(texMask, xx+iter*N_ATOMS);*/

	int val_bit = (fabsf(m_ix - m_jx) == 1 && jx>ix && distance2 < LIMIT_DISTANCE2)? 1:0;

	int reduced = blockReduce(val_bit);
	if(!ix) is_bumping_p[jx+index*N_ATOMS] = reduced;
}

__global__ void fragment_reduce(int* is_bumping, int* is_bumping_p){
	const int index = blockIdx.x;
	int x = threadIdx.x;
	int val_bit = is_bumping_p[x+index*N_ATOMS];
	int reduced = blockReduce(val_bit);
	if(!x) is_bumping[index] = (reduced)? 1:0;
}


__inline__ __device__ void warpReduce(int ind, int sho, int bum, int &ret1, int &ret2, int &ret3) {
	int im, sm, bm;
	for (int i = warpSize/2; i > 0; i/=2){
		im = __shfl_down_sync(0xffffffff, ind, i, 32);
		sm = __shfl_down_sync(0xffffffff, sho, i, 32);
		bm = __shfl_down_sync(0xffffffff, bum, i, 32);
		if(!(bm > bum || (bum==bm && sho>=sm))){
			ind = im;
			sho = sm;
			bum = bm;
		}
	}
	ret1=ind;	
	ret2=sho;
	ret3=bum;
}

__inline__ __device__ int find_best(int* shotgun, int* bumping, int index){
	int shot = shotgun[index];
	int bum = bumping[index];
	int ind = index;
	static __shared__ int sharedI[32];
	static __shared__ int sharedS[32];
	static __shared__ int sharedB[32];

	int lane = index % warpSize;
	int wid = index / warpSize;

	warpReduce(index, shot, bum, ind, shot, bum);

	if (lane==0){
		sharedI[wid]=ind;
		sharedS[wid]=shot;
		sharedB[wid]=bum;
	}
	
	__syncthreads();

	if(index < blockDim.x / warpSize){
		ind = sharedI[lane];
		bum = sharedB[lane];
		shot = sharedS[lane];
	} else {
		ind = 0;
		bum = 1;
		shot = 0;
	}
	
	if (wid==0) warpReduce(ind, shot, bum, ind, shot, bum);
	
	return ind;
}

__global__ void eval_angles(float* in, int* shotgun, int* bumping){
	
	__shared__ int best_angle;

	const int index = threadIdx.x;

	int best_index = find_best(shotgun, bumping, index);

	if(index == 0) {
		//printf("best: (%d: %f, %d, %d)\n", best_index, in[best_index*INSIZE], shotgun[best_index], bumping[best_index]);
		best_angle = best_index;
	}
	
	__syncthreads();

	//this line works assuming INSIZE<=MAX_ANGLE/precision. How do we remove this assumption? (probably need a for to copy multiple values)
	if(index < INSIZE) in[index] = in[best_angle*INSIZE+index];
}


#define cudaSafeCall(call)  \
        do {\
            hipError_t err = call;\
            if (hipSuccess != err) \
            {\
                printf("CUDA error in %s(%s): %s",__FILE__,__LINE__,hipGetErrorString(err));\
                exit(EXIT_FAILURE);\
            }\
        } while(0)

void cudaMemoryTest()
{
    const unsigned int N = 1048576;
    const unsigned int bytes = N * sizeof(int);
    int *h_a = (int*)malloc(bytes);
    int *d_a;
    cudaSafeCall(hipMalloc((int**)&d_a, bytes));

    memset(h_a, 0, bytes);
    cudaSafeCall(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost));
}

void ps_kern(float* in, float* out, float precision, float* score_pos, int* start, int* stop, int* mask)
{
	float *d_in, *d_score_pos;

	int *d_start, *d_stop, *d_mask, *d_shotgun;

	int *d_bumping, *d_bumping_partial;

	hipError_t status, status_cp, status_wb;
	hipStream_t s1, s2;
	hipEvent_t start_t, stop_t;

	status = hipMalloc((void**) &d_in, sizeof(float)*INSIZE*ceil(MAX_ANGLE/precision));
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status_cp = hipMemcpy(d_in, in, sizeof(float)*INSIZE, hipMemcpyHostToDevice);
	if(DEBUG && status_cp!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status_cp), __FILE__, __LINE__);

	status = hipMalloc((void**) &d_start, sizeof(int)*N_ATOMS);
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status_cp = hipMemcpy(d_start, start, sizeof(int)*N_ATOMS, hipMemcpyHostToDevice);
	if(DEBUG && status_cp!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status_cp), __FILE__, __LINE__);

	status = hipMalloc((void**) &d_stop, sizeof(int)*N_ATOMS);
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status_cp = hipMemcpy(d_stop, stop, sizeof(int)*N_ATOMS, hipMemcpyHostToDevice);
	if(DEBUG && status_cp!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status_cp), __FILE__, __LINE__);

	status = hipMalloc((void**)&d_bumping, sizeof(int)*ceil(MAX_ANGLE/precision));
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status = hipMalloc((void**)&d_bumping_partial, sizeof(int)*ceil(MAX_ANGLE/precision)*N_ATOMS);
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status = hipMalloc((void**)&d_shotgun, sizeof(int)*ceil(MAX_ANGLE/precision));
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status = hipMalloc((void**) &d_mask, sizeof(int)*MASKSIZE);
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status_cp = hipMemcpy(d_mask, mask, sizeof(int)*MASKSIZE, hipMemcpyHostToDevice);
	if(DEBUG && status_cp!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status_cp), __FILE__, __LINE__);

	status = hipMalloc((void**) &d_score_pos, sizeof(float)*VOLUMESIZE);
	if(DEBUG && status!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status), __FILE__, __LINE__);

	status_cp = hipMemcpy(d_score_pos, score_pos, sizeof(float)*VOLUMESIZE, hipMemcpyHostToDevice);
	if(DEBUG && status_cp!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status_cp), __FILE__, __LINE__);

	hipResourceDesc resDesc1;
	memset(&resDesc1, 0.0, sizeof(resDesc1));
	resDesc1.resType = hipResourceTypeLinear;
	resDesc1.res.linear.devPtr = d_score_pos;
	resDesc1.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc1.res.linear.desc.x = 32;
	resDesc1.res.linear.sizeInBytes = VOLUMESIZE*sizeof(float);

	hipResourceDesc resDesc2;
	memset(&resDesc2, 0, sizeof(resDesc2));
	resDesc2.resType = hipResourceTypeLinear;
	resDesc2.res.linear.devPtr = d_mask;
	resDesc2.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc2.res.linear.desc.x = 32;
	resDesc2.res.linear.sizeInBytes = MASKSIZE*sizeof(int);

	hipTextureDesc texDesc1;
	memset(&texDesc1, 0.0, sizeof(texDesc1));
	texDesc1.readMode = hipReadModeElementType;

	hipTextureDesc texDesc2;
	memset(&texDesc2, 0, sizeof(texDesc2));
	texDesc2.readMode = hipReadModeElementType;

	hipTextureObject_t texScore_pos=0;
	hipTextureObject_t texMask=0;
	hipCreateTextureObject(&texScore_pos, &resDesc1, &texDesc1, NULL);
	hipCreateTextureObject(&texMask, &resDesc2, &texDesc2, NULL);

	hipEventCreate(&start_t);
	hipEventCreate(&stop_t);

	hipStreamCreate(&s1);
	hipStreamCreate(&s2);

	dim3 bump_blocks(N_ATOMS,ceil(MAX_ANGLE/precision));

	hipEventRecord(start_t);

	/*cudaMemoryTest() calls and the function itself can be removed in the future, when we solved all the errors*/
	for (int i=0;i<N_FRAGS;++i){

		rotate<<<ceil(MAX_ANGLE/precision),N_ATOMS,0,s1>>>(d_in, d_mask, i, precision, d_start, d_stop);
		cudaMemoryTest();

		hipStreamSynchronize(s1);
		hipStreamSynchronize(s2);

		fragment_is_bumping<<<bump_blocks,N_ATOMS,0,s1>>>(d_in, d_mask, d_bumping_partial, i, precision);
		cudaMemoryTest();
		
		measure_shotgun<<<ceil(MAX_ANGLE/precision),N_ATOMS,0,s2>>>(d_in, d_score_pos, d_shotgun, precision, i);
		cudaMemoryTest();
		
		fragment_reduce<<<ceil(MAX_ANGLE/precision),N_ATOMS,0,s1>>>(d_bumping, d_bumping_partial);
		cudaMemoryTest();
		
		hipStreamSynchronize(s1);
		hipStreamSynchronize(s2);

		eval_angles<<<1,ceil(MAX_ANGLE/precision),0,s1>>>(d_in, d_shotgun, d_bumping);
		cudaMemoryTest();
	}

	hipDeviceSynchronize();

	hipEventRecord(stop_t);

	float milliseconds = 0;
	//Se chiamo la funzione qui sotto restituisce un errore cuda-memcheck: 
	//Program hit hipErrorInvalidHandle (error 400) due to "invalid resource handle" on CUDA API call to hipEventElapsedTime.
	//Wtf? Forse è colpa degli stream?
	hipEventElapsedTime(&milliseconds, start_t, stop_t);
	printf("\nKernels executed in %f milliseconds\n", milliseconds);

	status_wb = hipMemcpy(out, d_in, sizeof(float)*INSIZE, hipMemcpyDeviceToHost);
	if(DEBUG && status_wb!=hipSuccess)
		printf("%s in %s at line %d\n", hipGetErrorString(status_wb), __FILE__, __LINE__);

	hipDestroyTextureObject(texScore_pos);
	hipDestroyTextureObject(texMask);
	hipFree(d_bumping_partial);
	hipEventDestroy(start_t);
	hipEventDestroy(stop_t);
	hipStreamDestroy(s1);
	hipStreamDestroy(s2);
	hipFree(d_bumping);
	hipFree(d_shotgun);
	hipFree(d_start);
	hipFree(d_stop);
	hipFree(d_in);
}
